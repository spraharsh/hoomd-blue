
#include <hip/hip_runtime.h>
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: mphoward

/*!
 * \file mpcd/ParticleData.cu
 * \brief Defines GPU functions and kernels used by mpcd::ParticleData
 */

#ifdef ENABLE_MPI

#include "ParticleData.cuh"

#include "hoomd/extern/cub/cub/device/device_partition.cuh"
#include "hoomd/extern/cub/cub/thread/thread_load.cuh"

namespace mpcd
{
namespace gpu
{
namespace kernel
{
//! Kernel to partition particle data
/*!
 * \param d_out
 * \param mask
 * \param d_pos Device array of particle positions
 * \param d_vel Device array of particle velocities
 * \param d_tag Device array of particle tags
 * \param d_pos_alt Device array of particle positions (output)
 * \param d_vel_alt Device array of particle velocities (output)
 * \param d_tag_alt Device array of particle tags (output)
 * \param d_out Output array for packed particle data
 * \param d_comm_flags Communication flags (nonzero if particle should be migrated)
 * \param d_comm_flags_out Packed communication flags
 * \param d_scan Result of exclusive prefix sum
 * \param N Number of local particles
 *
 * Particles are removed by performing a selection using the result of an
 * exclusive prefix sum, stored in \a d_scan. The scan recovers the indexes
 * of the particles. A simple example illustrating the implementation follows:
 *
 * \verbatim
 * Particles:   0 1 2 3 4
 * Flags:       0|1 1|0 0
 * d_scan       0|0 1|2 2
 *              ---------
 * scan_keep:   0|1 1|1 2
 *              ---------
 * keep:        0,3,4 -> 0,1,2
 * remove:      1,2 -> 0,1
 * \endverbatim
 */
__global__ void remove_particles(mpcd::detail::pdata_element *d_out,
                                 const unsigned int n_keep,
                                 const Scalar4 *d_pos,
                                 const Scalar4 *d_vel,
                                 const unsigned int *d_tag,
                                 const unsigned int *d_comm_flags,
                                 Scalar4 *d_pos_alt,
                                 Scalar4 *d_vel_alt,
                                 unsigned int *d_tag_alt,
                                 unsigned int *d_comm_flags_alt,
                                 const unsigned int *d_scan,
                                 const unsigned int N)
    {
    const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= N) return;

    // read-only out of textures
    const unsigned int pid = cub::ThreadLoad<cub::LOAD_LDG>(d_scan + idx);
    const Scalar4 pos = cub::ThreadLoad<cub::LOAD_LDG>(d_pos + pid);
    const Scalar4 vel = cub::ThreadLoad<cub::LOAD_LDG>(d_vel + pid);
    const unsigned int tag = cub::ThreadLoad<cub::LOAD_LDG>(d_tag + pid);
    const unsigned int flag = cub::ThreadLoad<cub::LOAD_LDG>(d_comm_flags + pid);

    if (idx >= n_keep)
        {
        mpcd::detail::pdata_element p;
        p.pos = pos;
        p.vel = vel;
        p.tag = tag;
        p.comm_flag = flag;

        d_out[idx - n_keep] = p;
        }
    else
        {
        d_pos_alt[idx] = pos;
        d_vel_alt[idx] = vel;
        d_tag_alt[idx] = tag;
        d_comm_flags_alt[idx] = flag;
        }
    }

//! Kernel to transform communication flags for prefix sum
/*!
 * \param d_tmp Temporary storage to hold transformation (output)
 * \param d_comm_flags Communication flags
 * \param mask Bitwise mask for \a d_comm_flags
 * \param N Number of local particles
 *
 * Any communication flags that are bitwise AND with \a mask are transformed to
 * a 1 and stored in \a d_tmp.
 */
__global__ void mark_removed_particles(unsigned char *d_tmp_flag,
                                       unsigned int *d_tmp_id,
                                       const unsigned int *d_comm_flags,
                                       const unsigned int mask,
                                       const unsigned int N)
    {
    const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    // one thread per particle
    if (idx >= N) return;

    d_tmp_id[idx] = idx;
    d_tmp_flag[idx] = (d_comm_flags[idx] & mask) ? 0 : 1;
    }
} // end namespace kernel
} // end namespace gpu
} // end namespace mpcd


cudaError_t mpcd::gpu::mark_removed_particles(unsigned char *d_tmp_flag,
                                              unsigned int *d_tmp_id,
                                              const unsigned int *d_comm_flags,
                                              const unsigned int mask,
                                              const unsigned int N,
                                              const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        cudaFuncAttributes attr;
        cudaFuncGetAttributes(&attr, (const void*)mpcd::gpu::kernel::mark_removed_particles);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);
    dim3 grid(N / run_block_size + 1);
    mpcd::gpu::kernel::mark_removed_particles<<<grid, run_block_size>>>(d_tmp_flag,
                                                                        d_tmp_id,
                                                                        d_comm_flags,
                                                                        mask,
                                                                        N);
    return cudaSuccess;
    }

cudaError_t mpcd::gpu::partition_particles(void *d_tmp,
                                           size_t& tmp_bytes,
                                           const unsigned int *d_ids,
                                           const unsigned char *d_flags,
                                           unsigned int *d_out,
                                           unsigned int *d_num_select,
                                           const unsigned int N)
    {
    cub::DevicePartition::Flagged(d_tmp, tmp_bytes, d_ids, d_flags, d_out, d_num_select, N);
    return cudaSuccess;
    }

/*!
 * \param d_out Output array for packed particle data
 * \param mask Bitwise mask for \a d_comm_flags
 * \param d_pos Device array of particle positions
 * \param d_vel Device array of particle velocities
 * \param d_tag Device array of particle tags
 * \param d_comm_flags Device array of communication flags
 * \param d_pos_alt Device array of particle positions (output)
 * \param d_vel_alt Device array of particle velocities (output)
 * \param d_tag_alt Device array of particle tags (output)
 * \param d_comm_flags_alt Device array of communication flags (output)
 * \param d_scan Output from device scan of temporary flags
 * \param N Current number of particles
 *
 * \returns cudaSuccess on completion.
 */
cudaError_t mpcd::gpu::remove_particles(mpcd::detail::pdata_element *d_out,
                                        const unsigned int mask,
                                        const Scalar4 *d_pos,
                                        const Scalar4 *d_vel,
                                        const unsigned int *d_tag,
                                        const unsigned int *d_comm_flags,
                                        Scalar4 *d_pos_alt,
                                        Scalar4 *d_vel_alt,
                                        unsigned int *d_tag_alt,
                                        unsigned int *d_comm_flags_alt,
                                        unsigned int *d_scan,
                                        const unsigned int N)
    {
    // partition particle data into local and removed particles
    unsigned int block_size = 512;
    unsigned int n_blocks = N/block_size+1;

    mpcd::gpu::kernel::remove_particles<<<n_blocks, block_size>>>(d_out,
                                                                  mask,
                                                                  d_pos,
                                                                  d_vel,
                                                                  d_tag,
                                                                  d_comm_flags,
                                                                  d_pos_alt,
                                                                  d_vel_alt,
                                                                  d_tag_alt,
                                                                  d_comm_flags_alt,
                                                                  d_scan,
                                                                  N);
    return cudaSuccess;
    }


namespace mpcd
{
namespace gpu
{
namespace kernel
{
//! Kernel to partition particle data
/*!
 * \param old_nparticles old local particle count
 * \param num_add_ptls Number of particles in input array
 * \param d_pos Device array of particle positions
 * \param d_vel Device array of particle velocities
 * \param d_tag Device array of particle tags
 * \param d_comm_flags Device array of communication flags
 * \param d_in Device array of packed input particle data
 * \param mask Bitwise mask for received particles to unmask
 *
 * Particle data is appended to the end of the particle data arrays from the
 * packed buffer. Communication flags of new particles are unmasked.
 */
__global__ void add_particles(unsigned int old_nparticles,
                              unsigned int num_add_ptls,
                              Scalar4 *d_pos,
                              Scalar4 *d_vel,
                              unsigned int *d_tag,
                              unsigned int *d_comm_flags,
                              const mpcd::detail::pdata_element *d_in,
                              const unsigned int mask)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= num_add_ptls) return;

    mpcd::detail::pdata_element p = d_in[idx];

    unsigned int add_idx = old_nparticles + idx;
    d_pos[add_idx] = p.pos;
    d_vel[add_idx] = p.vel;
    d_tag[add_idx] = p.tag;
    d_comm_flags[add_idx] = p.comm_flag & ~mask;
    }
} // end namespace kernel
} // end namespace gpu
} // end namespace mpcd

/*!
 * \param old_nparticles old local particle count
 * \param num_add_ptls Number of particles in input array
 * \param d_pos Device array of particle positions
 * \param d_vel Device array of particle velocities
 * \param d_tag Device array of particle tags
 * \param d_comm_flags Device array of communication flags
 * \param d_in Device array of packed input particle data
 * \param mask Bitwise mask for received particles to unmask
 *
 * Particle data is appended to the end of the particle data arrays from the
 * packed buffer. Communication flags of new particles are unmasked.
 */
void mpcd::gpu::add_particles(unsigned int old_nparticles,
                              unsigned int num_add_ptls,
                              Scalar4 *d_pos,
                              Scalar4 *d_vel,
                              unsigned int *d_tag,
                              unsigned int *d_comm_flags,
                              const mpcd::detail::pdata_element *d_in,
                              const unsigned int mask)
    {
    unsigned int block_size = 512;
    unsigned int n_blocks = num_add_ptls/block_size + 1;

    mpcd::gpu::kernel::add_particles<<<n_blocks, block_size>>>(old_nparticles,
                                                               num_add_ptls,
                                                               d_pos,
                                                               d_vel,
                                                               d_tag,
                                                               d_comm_flags,
                                                               d_in,
                                                               mask);
    }

#endif // ENABLE_MPI
